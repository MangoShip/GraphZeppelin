// This program computes the sum of two vectors of length N
// By: Nick from CoffeeBeforeArch


#include <hip/hip_runtime.h>
#include <algorithm>
#include <cassert>
#include <iostream>
#include <vector>

__device__ int mutex = 0;

// CUDA kernel for vector addition
// __global__ means this is called from the CPU, and runs on the GPU
__global__ void vectorAdd(const int *__restrict a, const int *__restrict b,
                          int *__restrict c, int N) {
  // Calculate global thread ID
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  // Boundary check
  if (tid < N) {

    bool blocked = true; 

    while(blocked) {
      if(0 == (atomicCAS(&mutex, 0, 1))) {
        c[tid] = a[tid] + b[tid];
        atomicExch(&mutex, 0);
        break;
      }
    }
  }
}

// Check vector add result
void verify_result(std::vector<int> &a, std::vector<int> &b,
                   std::vector<int> &c) {
  for (int i = 0; i < a.size(); i++) {
    assert(c[i] == a[i] + b[i]);
  }
}

int main() {
  // Array size of 2^16 (65536 elements)
  constexpr int N = 10;
  constexpr size_t bytes = sizeof(int) * N;

  // Vectors for holding the host-side (CPU-side) data
  std::vector<int> a;
  a.reserve(N);
  std::vector<int> b;
  b.reserve(N);
  std::vector<int> c;
  c.reserve(N);

  // Initialize random numbers in each array
  for (int i = 0; i < N; i++) {
    a.push_back(1);
    b.push_back(2);
  }

  // Allocate memory on the device
  int *d_a, *d_b, *d_c;
  //int *mutex;
  hipMalloc(&d_a, bytes);
  hipMalloc(&d_b, bytes);
  hipMalloc(&d_c, bytes);
  //cudaMallocManaged(&mutex, sizeof(int));

  // Copy data from the host to the device (CPU -> GPU)
  hipMemcpy(d_a, a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(d_b, b.data(), bytes, hipMemcpyHostToDevice);
  //mutex[0] = 0;

  // Threads per CTA (1024)
  int NUM_THREADS = 1 << 10;

  // CTAs per Grid
  // We need to launch at LEAST as many threads as we have elements
  // This equation pads an extra CTA to the grid if N cannot evenly be divided
  // by NUM_THREADS (e.g. N = 1025, NUM_THREADS = 1024)
  int NUM_BLOCKS = (N + NUM_THREADS - 1) / NUM_THREADS;

  // Launch the kernel on the GPU
  // Kernel calls are asynchronous (the CPU program continues execution after
  // call, but no necessarily before the kernel finishes)
  vectorAdd<<<NUM_BLOCKS, NUM_THREADS>>>(d_a, d_b, d_c, N);

  // Copy sum vector from device to host
  // cudaMemcpy is a synchronous operation, and waits for the prior kernel
  // launch to complete (both go to the default stream in this case).
  // Therefore, this cudaMemcpy acts as both a memcpy and synchronization
  // barrier.
  hipMemcpy(c.data(), d_c, bytes, hipMemcpyDeviceToHost);

  // Check result for errors
  verify_result(a, b, c);

  // Free memory on device
  hipFree(d_a);
  hipFree(d_b);
  hipFree(d_c);

  std::cout << "COMPLETED SUCCESSFULLY\n";

  return 0;
}