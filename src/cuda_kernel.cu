#include "hip/hip_runtime.h"

#include <vector>
#include <cuda_xxhash64.cuh>
#include <graph.h>
#include <sketch.h>
#include "../include/cuda_kernel.cuh"

typedef unsigned long long int uint64_cu;
typedef uint64_cu vec_t_cu;

/*
*   
*   Bucket Functions
*
*/

// Source: http://graphics.stanford.edu/~seander/bithacks.html#ZerosOnRightLinear
__device__ int ctzll(col_hash_t v) {
  int c;
  if (v) {
    v = (v ^ (v - 1)) >> 1;
    for (c = 0; v; c++) {
      v >>= 1;
    }
  }
  else {
    c = 8 * sizeof(v);
  }
  return c;
}

__device__ col_hash_t bucket_get_index_depth(const vec_t_cu update_idx, const long seed_and_col, const vec_hash_t max_depth) {
  // Update CUDA_XXH, confirm they are correct with xxhash_test.cu
  col_hash_t depth_hash = CUDA_XXH64(&update_idx, sizeof(vec_t), seed_and_col);
  depth_hash |= (1ull << max_depth); // assert not > max_depth by ORing

  return ctzll(depth_hash);
}

__device__ vec_hash_t bucket_get_index_hash(const vec_t update_idx, const long sketch_seed) {
  return CUDA_XXH64(&update_idx, sizeof(vec_t), sketch_seed);
}

__device__ bool bucket_is_good(const vec_t a, const vec_hash_t c, const long sketch_seed) {
  return c == bucket_get_index_hash(a, sketch_seed);
}

__device__ void bucket_update(vec_t_cu& a, vec_hash_t& c, const vec_t_cu& update_idx, const vec_hash_t& update_hash) {
  atomicXor(&a, update_idx);
  atomicXor(&c, update_hash);
}

__device__ edge_id_t device_concat_pairing_fn(node_id_t i, node_id_t j) {
  // swap i,j if necessary
  if (i > j) {
    node_id_t temp = i;
    i = j;
    j = temp;
  }
  return ((edge_id_t)i << (sizeof(node_id_t) * 8)) | j;
}


/*
*   
*   Sketch's Update Functions
*
*/

__global__ void gtsStream_kernel(int stream_id, volatile int *edgeWriteEnabled, node_id_t src, vec_t* edgeUpdates, vec_t prev_offset, size_t update_size, node_id_t num_nodes,
    int num_sketches, size_t num_elems, size_t num_columns, size_t num_guesses, CudaSketch* cudaSketches, long* sketchSeeds) {
      
  extern __shared__ vec_t_cu sketches[];
  vec_t_cu* bucket_a = sketches;
  vec_hash_t* bucket_c = (vec_hash_t*)&bucket_a[num_elems * num_sketches];

  // Each thread will initialize a bucket
  for (int i = threadIdx.x; i < num_sketches * num_elems; i += blockDim.x) {
    bucket_a[i] = 0;
    bucket_c[i] = 0;
  }

  __syncthreads();

  // Update node's sketches
  for (int id = threadIdx.x; id < update_size + num_sketches; id += blockDim.x) {
    
    int sketch_offset = id % num_sketches;
    int update_offset = ((id / num_sketches) * num_sketches);
    
    for (int i = 0; i < num_sketches; i++) {

      if ((prev_offset + update_offset + i) >= prev_offset + update_size) {
        break;
      }

      vec_hash_t checksum = bucket_get_index_hash(edgeUpdates[prev_offset + update_offset + i], sketchSeeds[(src * num_sketches) + sketch_offset]);

      // Update depth 0 bucket
      bucket_update(bucket_a[(sketch_offset * num_elems) + num_elems - 1], bucket_c[(sketch_offset * num_elems) + num_elems - 1], edgeUpdates[prev_offset + update_offset + i], checksum);

      // Update higher depth buckets
      for (unsigned j = 0; j < num_columns; ++j) {
        col_hash_t depth = bucket_get_index_depth(edgeUpdates[prev_offset + update_offset + i], sketchSeeds[(src * num_sketches) + sketch_offset] + j*5, num_guesses);
        size_t bucket_id = j * num_guesses + depth;
        if(depth < num_guesses)
          bucket_update(bucket_a[(sketch_offset * num_elems) + bucket_id], bucket_c[(sketch_offset * num_elems) + bucket_id], edgeUpdates[prev_offset + update_offset + i], checksum);
      }
    }
  }

  __syncthreads();

  // Each thread will trasfer a bucket back to global memory
  for (int i = threadIdx.x; i < num_sketches * num_elems; i += blockDim.x) {
      int sketch_offset = i / num_elems; 
      int elem_id = i % num_elems;

      CudaSketch curr_cudaSketch = cudaSketches[(src * num_sketches) + sketch_offset];

      vec_t_cu* curr_bucket_a = (vec_t_cu*)curr_cudaSketch.d_bucket_a;
      vec_hash_t* curr_bucket_c = curr_cudaSketch.d_bucket_c;

      atomicXor(&curr_bucket_a[elem_id], bucket_a[i]);
      atomicXor(&curr_bucket_c[elem_id], bucket_c[i]);
  }

  __syncthreads();
  if (threadIdx.x == 0) {
    edgeWriteEnabled[stream_id] = 1;
  }
  
}

// Version 6: Kernel code of handling all the stream updates
// Two threads will be responsible for one edge update -> one thread is only modifying one node's sketches.
// Placing sketches in shared memory, each thread is doing log n updates on one slice of sketch.
// Applying newest verison of sketch update function
__global__ void doubleStream_update(vec_t* edgeUpdates, int* nodeNumUpdates, vec_t* nodeStartIndex, node_id_t num_nodes,
    int num_sketches, size_t num_elems, size_t num_columns, size_t num_guesses, CudaSketch* cudaSketches, long* sketchSeeds) {

  if (blockIdx.x < num_nodes){
    
    extern __shared__ vec_t_cu sketches[];
    vec_t_cu* bucket_a = sketches;
    vec_hash_t* bucket_c = (vec_hash_t*)&bucket_a[num_elems * num_sketches];
    int node = blockIdx.x;
    vec_t startIndex = nodeStartIndex[node];

    // Each thread will initialize a bucket
    for (int i = threadIdx.x; i < num_sketches * num_elems; i += blockDim.x) {
      bucket_a[i] = 0;
      bucket_c[i] = 0;
    }

    __syncthreads();

    // Update node's sketches
    for (int id = threadIdx.x; id < nodeNumUpdates[node] + num_sketches; id += blockDim.x) {
      
      int sketch_offset = id % num_sketches;
      int update_offset = ((id / num_sketches) * num_sketches);
      
      for (int i = 0; i < num_sketches; i++) {

        if ((startIndex + update_offset + i) >= startIndex + nodeNumUpdates[node]) {
          break;
        }

        vec_hash_t checksum = bucket_get_index_hash(edgeUpdates[startIndex + update_offset + i], sketchSeeds[(node * num_sketches) + sketch_offset]);

        // Update depth 0 bucket
        bucket_update(bucket_a[(sketch_offset * num_elems) + num_elems - 1], bucket_c[(sketch_offset * num_elems) + num_elems - 1], edgeUpdates[startIndex + update_offset + i], checksum);

        // Update higher depth buckets
        for (unsigned j = 0; j < num_columns; ++j) {
          col_hash_t depth = bucket_get_index_depth(edgeUpdates[startIndex + update_offset + i], sketchSeeds[(node * num_sketches) + sketch_offset] + j*5, num_guesses);
          size_t bucket_id = j * num_guesses + depth;
          if(depth < num_guesses)
            bucket_update(bucket_a[(sketch_offset * num_elems) + bucket_id], bucket_c[(sketch_offset * num_elems) + bucket_id], edgeUpdates[startIndex + update_offset + i], checksum);
        }
      }
    }

    __syncthreads();

    // Each thread will trasfer a bucket back to global memory
    for (int i = threadIdx.x; i < num_sketches * num_elems; i += blockDim.x) {
        int sketch_offset = i / num_elems; 
        int elem_id = i % num_elems;

        CudaSketch curr_cudaSketch = cudaSketches[(node * num_sketches) + sketch_offset];

        vec_t_cu* curr_bucket_a = (vec_t_cu*)curr_cudaSketch.d_bucket_a;
        vec_hash_t* curr_bucket_c = curr_cudaSketch.d_bucket_c;

        curr_bucket_a[elem_id] = bucket_a[i];
        curr_bucket_c[elem_id] = bucket_c[i];
        
    }
    __syncthreads();
  }
}

// Function that calls sketch update kernel code.
void CudaKernel::gtsStreamUpdate(int num_threads, int num_blocks, int stream_id, node_id_t src, hipStream_t stream, vec_t prev_offset, size_t update_size, CudaUpdateParams* cudaUpdateParams, CudaSketch* cudaSketches, long* sketchSeeds) {
  // Unwarp variables from cudaUpdateParams
  vec_t *edgeUpdates = cudaUpdateParams[0].edgeUpdates;
  volatile int *edgeWriteEnabled = cudaUpdateParams[0].edgeWriteEnabled;

  node_id_t num_nodes = cudaUpdateParams[0].num_nodes;
  
  int num_sketches = cudaUpdateParams[0].num_sketches;

  size_t num_elems = cudaUpdateParams[0].num_elems;
  size_t num_columns = cudaUpdateParams[0].num_columns;
  size_t num_guesses = cudaUpdateParams[0].num_guesses;

  int maxbytes = num_elems * num_sketches * sizeof(vec_t_cu) + num_elems * num_sketches * sizeof(vec_hash_t);

  hipFuncSetAttribute(reinterpret_cast<const void*>(gtsStream_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
  gtsStream_kernel<<<num_blocks, num_threads, maxbytes, stream>>>(stream_id, edgeWriteEnabled, src, edgeUpdates, prev_offset, update_size, num_nodes, num_sketches, num_elems, num_columns, num_guesses, cudaSketches, sketchSeeds);
}

// Function that calls stream update kernel code.
/*void CudaKernel::streamUpdate(int num_threads, int num_blocks, CudaUpdateParams* cudaUpdateParams, CudaSketch* cudaSketches, long* sketchSeeds) {

  // Unwarp variables from cudaUpdateParams
  vec_t *edgeUpdates = cudaUpdateParams[0].edgeUpdates;
  int *nodeNumUpdates = cudaUpdateParams[0].nodeNumUpdates;
  vec_t *nodeStartIndex = cudaUpdateParams[0].nodeStartIndex;

  node_id_t num_nodes = cudaUpdateParams[0].num_nodes;

  int num_sketches = cudaUpdateParams[0].num_sketches;
  
  size_t num_elems = cudaUpdateParams[0].num_elems;
  size_t num_columns = cudaUpdateParams[0].num_columns;
  size_t num_guesses = cudaUpdateParams[0].num_guesses;

  int maxbytes = num_elems * num_sketches * sizeof(vec_t_cu) + num_elems * num_sketches * sizeof(vec_hash_t);
  
  // Increase maximum of dynamic shared memory size
  // Note: Only works if GPU has enough shared memory capacity to store sketches for each vertex
  hipFuncSetAttribute(reinterpret_cast<const void*>(doubleStream_update), hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);

  /*
      Note (Only when using shared memory): I have noticed that unwrapping variables within kernel code
      caused these parameter variables to stay within global memory, creating more latency. Therefore, unwrapping these 
      variables then passing as argument of the kernel code avoids that issue.
  */ 
  /*doubleStream_update<<<num_blocks, num_threads, maxbytes>>>(edgeUpdates, nodeNumUpdates, nodeStartIndex, num_nodes, num_sketches, num_elems, num_columns, num_guesses, cudaSketches, sketchSeeds);

  hipDeviceSynchronize();
}*/

/*
*   
*   Sketch's Query Functions
*
*/

__device__ Edge cuda_inv_concat_pairing_fn(uint64_t idx) {
  uint8_t num_bits = sizeof(node_id_t) * 8;
  node_id_t j = idx & 0xFFFFFFFF;
  node_id_t i = idx >> num_bits;
  return {i, j};
}

__device__ node_id_t get_parent(node_id_t* parent, node_id_t node) {
  if (parent[node] == node) return node;
  return parent[node] = get_parent(parent, parent[node]);
}

__global__ void sketch_query(node_id_t* reps, CudaQuery* query, size_t* sample_idxs, node_id_t num_nodes, int num_sketches, size_t num_elems, size_t num_columns, size_t num_guesses, CudaSketch* cudaSketches) {

  // Get thread id
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (tid < num_nodes) {
    int query_id = reps[tid];
    CudaSketch cudaSketch = cudaSketches[(query_id * num_sketches) + sample_idxs[query_id]];

    vec_t_cu* bucket_a = (vec_t_cu*)cudaSketch.d_bucket_a;
    vec_hash_t* bucket_c = cudaSketch.d_bucket_c;

    if (bucket_a[num_elems - 1] == 0 && bucket_c[num_elems - 1] == 0) {
      query[query_id] = {cuda_inv_concat_pairing_fn(0), ZERO}; // the "first" bucket is deterministic so if all zero then no edges to return
      return;     
    }

    if (bucket_is_good(bucket_a[num_elems - 1], bucket_c[num_elems - 1], cudaSketch.seed)) {
      query[query_id] = {cuda_inv_concat_pairing_fn(bucket_a[num_elems - 1]), GOOD};
      return;      
    }

    for (unsigned i = 0; i < num_columns; ++i) {
      for (unsigned j = 0; j < num_guesses; ++j) {
        unsigned bucket_id = i * num_guesses + j;
        if (bucket_is_good(bucket_a[bucket_id], bucket_c[bucket_id], cudaSketch.seed)) {
          query[query_id] = {cuda_inv_concat_pairing_fn(bucket_a[bucket_id]), GOOD};
          return;          
        }
      }
    }
    query[query_id] = {cuda_inv_concat_pairing_fn(0), FAIL};
  }
}

void CudaKernel::cuda_sample_supernodes(int num_threads, int num_blocks, CudaCCParams* cudaCCParams, CudaSketch* cudaSketches) {
  // Unwarp variables from cudaCCParams
  node_id_t* reps = cudaCCParams[0].reps;
  CudaQuery* query = cudaCCParams[0].query;
  size_t* sample_idxs = cudaCCParams[0].sample_idxs;

  node_id_t num_nodes = cudaCCParams[0].num_nodes[0];

  int num_sketches = cudaCCParams[0].num_sketches;

  size_t num_elems = cudaCCParams[0].num_elems;
  size_t num_columns = cudaCCParams[0].num_columns;
  size_t num_guesses = cudaCCParams[0].num_guesses;

  // Call query kernel
  sketch_query<<<num_blocks, num_threads>>>(reps, query, sample_idxs, num_nodes, num_sketches, num_elems, num_columns, num_guesses, cudaSketches);

  hipDeviceSynchronize();
}

__global__ void supernodes_to_merge(node_id_t* reps, node_id_t* temp_reps, CudaQuery* query, node_id_t* parent, node_id_t* size, CudaToMerge* to_merge, node_id_t* num_nodes, bool* modified) {
  // Get thread id
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  // Note: Have 1 thread to handle all workload (Temporary)
  if (tid == 0) {
    int temp_reps_id = 0;

    for (int i = 0; i < num_nodes[0]; i++) {
      int query_id = reps[i];

      // unpack query result
      Edge edge = query[query_id].edge;
      SampleSketchRet ret_code = query[query_id].ret_code;

      if (ret_code == ZERO) {
        continue;
      }
      else if (ret_code == FAIL) {
        modified[0] = true;
        temp_reps[temp_reps_id] = query_id;
        temp_reps_id++;
        continue;
      }
      else { // ret_code == GOOD
        // query dsu
        node_id_t a = get_parent(parent, edge.src);
        node_id_t b = get_parent(parent, edge.dst);
        if (a == b) continue;

        // make a the parent of b
        if (size[a] < size[b]) {
          node_id_t temp = a;
          a = b;
          b = temp;
        }
        parent[b] = a;
        size[a] += size[b];

        // add b and any of the nodes to merge with it to a's vector
        CudaToMerge a_merge = to_merge[a];
        CudaToMerge b_merge = to_merge[b];

        a_merge.children[a_merge.size[0]] = b;
        a_merge.size[0]++;

        // Fill b's children to a
        for (int j = 0; j < b_merge.size[0]; j++) {
          a_merge.children[a_merge.size[0]] = b_merge.children[j];
          a_merge.size[0]++;
          b_merge.children[j] = 0;
        }

        // Clear b
        b_merge.size[0] = 0;
        modified[0] = true;
      }
    }

    // remove nodes added to new_reps due to sketch failures that
    // did end up being able to merge after all
    int temp_reps_size = temp_reps_id;
    int reps_id = 0;

    for (int i = 0; i < temp_reps_size; i++) {
      node_id_t a = temp_reps[i];
      if (to_merge[a].size[0] == 0) {
        reps[reps_id] = a;
        reps_id++;
      }
    }

    for (int i = 0; i < num_nodes[1]; i++) {
      if (to_merge[i].size[0] != 0) {
        reps[reps_id] = i;
        reps_id++;
      }
    }

    num_nodes[0] = reps_id;
  }
}

void CudaKernel::cuda_supernodes_to_merge(int num_threads, int num_blocks, CudaCCParams* cudaCCParams) {
  // Unwarp variables from cudaCCParams
  node_id_t* reps = cudaCCParams[0].reps;
  node_id_t* temp_reps = cudaCCParams[0].temp_reps;
  CudaQuery* query = cudaCCParams[0].query;

  node_id_t* parent = cudaCCParams[0].parent;
  node_id_t* size = cudaCCParams[0].size;

  CudaToMerge* to_merge = cudaCCParams[0].to_merge;

  node_id_t* num_nodes = cudaCCParams[0].num_nodes;

  bool* modified = cudaCCParams[0].modified;

  // Call supernodes_to_merge kernel
  supernodes_to_merge<<<num_blocks, num_threads>>>(reps, temp_reps, query, parent, size, to_merge, num_nodes, modified);
  hipDeviceSynchronize();
}

__global__ void merge_supernodes(node_id_t* reps, CudaToMerge* to_merge, node_id_t* num_nodes, size_t* sample_idxs, size_t* merged_sketches, int num_sketches, size_t num_elems, CudaSketch* cudaSketches) {
  // Get thread id
  int tid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (tid < num_nodes[0]) {
    node_id_t a = reps[tid];

    // perform merging of nodes b into node a
    for (int i = 0; i < to_merge[a].size[0]; i++) {
      node_id_t b = to_merge[a].children[i];

      if (sample_idxs[b] > sample_idxs[a]) {
        sample_idxs[a] = sample_idxs[b];
      }
      if (merged_sketches[b] < merged_sketches[a]) {
        merged_sketches[a] = merged_sketches[b];
      }
      
      // Merge sketches
      for (int j = sample_idxs[a]; j < merged_sketches[a]; ++j) {
        CudaSketch cudaSketch1 = cudaSketches[(a * num_sketches) + j];
        CudaSketch cudaSketch2 = cudaSketches[(b * num_sketches) + j];

        vec_t_cu* sketch1_bucket_a = (vec_t_cu*)cudaSketch1.d_bucket_a;
        vec_hash_t* sketch1_bucket_c = cudaSketch1.d_bucket_c;

        vec_t_cu* sketch2_bucket_a = (vec_t_cu*)cudaSketch2.d_bucket_a;
        vec_hash_t* sketch2_bucket_c = cudaSketch2.d_bucket_c;

        if(sketch2_bucket_a[num_elems - 1] == 0 && sketch2_bucket_c[num_elems - 1] == 0) {
          continue;
        }
        for (int k = 0; k < num_elems; k++) {
          sketch1_bucket_a[k] ^= sketch2_bucket_a[k];
          sketch1_bucket_c[k] ^= sketch2_bucket_c[k];
        }
      }
  
    }
  }
}

void CudaKernel::cuda_merge_supernodes(int num_threads, int num_blocks, CudaCCParams* cudaCCParams, CudaSketch* cudaSketches) {
  // Unwarp variables from cudaCCParams
  node_id_t* reps = cudaCCParams[0].reps;

  CudaToMerge* to_merge = cudaCCParams[0].to_merge;

  node_id_t* num_nodes = cudaCCParams[0].num_nodes;

  size_t* sample_idxs = cudaCCParams[0].sample_idxs;
  size_t* merged_sketches = cudaCCParams[0].merged_sketches;

  int num_sketches = cudaCCParams[0].num_sketches;

  size_t num_elems = cudaCCParams[0].num_elems;

  // Call supernodes_to_merge kernel
  merge_supernodes<<<num_blocks, num_threads>>>(reps, to_merge, num_nodes, sample_idxs, merged_sketches, num_sketches, num_elems, cudaSketches);
  hipDeviceSynchronize();
}