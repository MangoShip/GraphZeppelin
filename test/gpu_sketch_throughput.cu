#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <vector>

#include <sketch.h>
#include "../src/cuda_kernel.cu"

static size_t get_seed() {
  auto now = std::chrono::high_resolution_clock::now();
  return std::chrono::duration_cast<std::chrono::nanoseconds>(now.time_since_epoch()).count();
}

__global__ void gpuSketchTest_kernel(int num_device_blocks, node_id_t num_nodes, size_t num_updates, size_t num_buckets, Bucket* buckets, size_t num_columns, size_t bkt_per_col, size_t sketchSeed) {

  extern __shared__ vec_t_cu sketches[];
  vec_t_cu* bucket_a = sketches;
  vec_hash_t* bucket_c = (vec_hash_t*)&bucket_a[num_buckets];

  for (size_t i = threadIdx.x; i < num_buckets; i += blockDim.x) {
    bucket_a[i] = 0;
    bucket_c[i] = 0;
  }

  __syncthreads();

  size_t update_offset = num_updates * num_columns * blockIdx.x;
  node_id_t node_id = blockIdx.x / num_nodes;
  for (size_t id = threadIdx.x; id < num_updates * num_columns; id += blockDim.x) {

    size_t column_id = (update_offset + id) % num_columns;
    size_t update_id = (update_offset + id) / num_columns;

    // Get random edge id based on current update_id
    //vec_t edge_id = update_id % num_nodes;
    vec_t edge_id = device_concat_pairing_fn(node_id, update_id % num_nodes);

    vec_hash_t checksum = bucket_get_index_hash(edge_id, sketchSeed);
    
    if ((blockIdx.x == num_device_blocks - 1)  && (column_id == 0)) {
      // Update depth 0 bucket
      bucket_update(bucket_a[num_buckets - 1], bucket_c[num_buckets - 1], edge_id, checksum);
    }

    // Update higher depth buckets
    col_hash_t depth = bucket_get_index_depth(edge_id, sketchSeed + ((column_id) * 5), bkt_per_col);
    size_t bucket_id = column_id * bkt_per_col + depth;
    if(depth < bkt_per_col)
      bucket_update(bucket_a[bucket_id], bucket_c[bucket_id], edge_id, checksum);
  }

  __syncthreads();

  for (size_t i = threadIdx.x; i < num_buckets; i += blockDim.x) {
    atomicXor((vec_t_cu*)&buckets[(node_id * num_buckets) + i].alpha, bucket_a[i]);
    atomicXor((vec_t_cu*)&buckets[(node_id * num_buckets) + i].gamma, (vec_t_cu)bucket_c[i]);
  }
}


int main(int argc, char **argv) {
  if (argc != 3) {
    std::cout << "ERROR: Incorrect number of arguments!" << std::endl;
    std::cout << "Arguments: num_nodes num_updates" << std::endl;
    exit(EXIT_FAILURE);
  }

  std::cout << "SKETCH COMPUTE THROUGHPUT TEST - GPU:\n";

  int device_id = hipGetDevice(&device_id);
  int device_count = 0;
  hipGetDeviceCount(&device_count);
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device_id);
  std::cout << "-----CUDA Device Information-----\n";
  std::cout << "CUDA Device Count: " << device_count << "\n";
  std::cout << "CUDA Device ID: " << device_id << "\n";
  std::cout << "CUDA Device Number of SMs: " << deviceProp.multiProcessorCount << "\n"; 
  std::cout << "CUDA Max. Shared memory per Block: " << (double)deviceProp.sharedMemPerBlockOptin / 1000 << "KB\n";

  size_t free_memory;
  size_t total_memory;

  hipMemGetInfo(&free_memory, &total_memory);
  std::cout << "GPU Free (Available) Memory: " << (double)free_memory / 1000000000 << "GB\n";
  std::cout << "GPU Total Memory: " << (double)total_memory / 1000000000 << "GB\n";
  std::cout << "\n";

  node_id_t num_nodes = std::atoi(argv[1]);
  size_t num_updates = std::stoull(argv[2]);

  // Single Sketch that fills up entire GPU memory
  /*SketchParams sketchParams;
  sketchParams.bkt_per_col = Sketch::calc_bkt_per_col(Sketch::calc_vector_length(num_nodes));
  sketchParams.num_columns = (free_memory / sizeof(Bucket)) / sketchParams.bkt_per_col;
  sketchParams.num_columns -= 100000; // Reduce some columns so not using 100% GPU memory (still using nearly 100%)
  //sketchParams.num_columns /= 4;
  sketchParams.num_buckets = sketchParams.num_columns * sketchParams.bkt_per_col + 1;*/
  
  // Single Sketch with size corresponding to num_nodes
  SketchParams sketchParams;
  sketchParams.num_samples = Sketch::calc_cc_samples(num_nodes, 1);
  sketchParams.num_columns = sketchParams.num_samples * Sketch::default_cols_per_sample;
  sketchParams.bkt_per_col = Sketch::calc_bkt_per_col(Sketch::calc_vector_length(num_nodes));
  sketchParams.num_buckets = sketchParams.num_columns * sketchParams.bkt_per_col + 1;

  // Single Sketch with size corresponding to GPU SM
  /*SketchParams sketchParams;
  sketchParams.num_columns = deviceProp.multiProcessorCount * 64;
  sketchParams.bkt_per_col = Sketch::calc_bkt_per_col(Sketch::calc_vector_length(num_nodes));
  sketchParams.num_buckets = sketchParams.num_columns * sketchParams.bkt_per_col + 1;*/

  std::cout << "-----Sketch Information-----\n";
  std::cout << "num_nodes: " << num_nodes << "\n";
  std::cout << "num_updates: " << num_updates << "\n";
  std::cout << "bkt_per_col: " << sketchParams.bkt_per_col << "\n";
  std::cout << "num_columns: " << sketchParams.num_columns << "\n";
  std::cout << "num_buckets: " << sketchParams.num_buckets << "\n";
  std::cout << "\n";

  int num_device_threads = 1024;
  int num_updates_per_blocks = (sketchParams.num_buckets * sizeof(Bucket)) / sizeof(node_id_t);
  int num_device_blocks = std::ceil((double)num_updates / num_updates_per_blocks);

  std::cout << "Batch Size: " << num_updates_per_blocks << "\n\n";

  /*int *num_tb_columns;
  gpuErrchk(hipMallocManaged(&num_tb_columns, num_device_blocks * sizeof(int)));

  for (int i = 0; i < num_device_blocks ; i++) {
    num_tb_columns[i] = sketchParams.num_columns / num_device_blocks;
  }

  // If num_columns doesn't get divided evenly
  size_t leftover_num_columns = sketchParams.num_columns - ((sketchParams.num_columns / num_device_blocks) * num_device_blocks);
  int k_id = num_device_blocks - 1;
  while (leftover_num_columns > 0) {
    num_tb_columns[k_id]++;
    k_id--;
    leftover_num_columns--;
  }*/

  //size_t num_columns_per_block = (deviceProp.sharedMemPerBlockOptin / sizeof(Bucket)) / sketchParams.bkt_per_col;
  /*size_t num_columns_per_block = 64;
  /size_t num_buckets_per_block = num_columns_per_block * sketchParams.bkt_per_col + 1; 
  size_t maxBytes = num_buckets_per_block * sizeof(vec_t_cu) + num_buckets_per_block * sizeof(vec_hash_t);*/

  //size_t num_last_tb_buckets = (num_tb_columns[num_device_blocks - 1] * sketchParams.bkt_per_col) + 1;
  //size_t maxBytes = (num_last_tb_buckets * sizeof(vec_t_cu)) + (num_last_tb_buckets * sizeof(vec_hash_t));
  size_t maxBytes = (sketchParams.num_buckets * sizeof(vec_t_cu)) + (sketchParams.num_buckets * sizeof(vec_hash_t));
  hipFuncSetAttribute(reinterpret_cast<const void*>(gpuSketchTest_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, maxBytes);

  std::cout << "-----GPU Kernel Information-----\n";
  std::cout << "Number of thread blocks: " << num_device_blocks << "\n";
  std::cout << "Number of threads per block: " << num_device_threads << "\n";
  std::cout << "Memory Size for buckets: " << (double)(num_nodes * sketchParams.num_buckets * sizeof(Bucket)) / 1000000000 << "GB\n";
  //std::cout << "Number of columns per thread block: " << num_columns_per_block << "\n";
  /*std::cout << "Number of columns of each thread block: ";
  for (int i = 0; i < num_device_blocks ; i++) {
    std::cout << num_tb_columns[i] << ", ";
  }
  std::cout << "\n";*/
  std::cout << "  Allocated Shared Memory of: " << (double)maxBytes / 1000 << "KB\n";
  std::cout << "\n";

  Bucket* d_buckets;
  gpuErrchk(hipMalloc(&d_buckets, num_nodes * sketchParams.num_buckets * sizeof(Bucket)));

  size_t sketchSeed = get_seed();

  auto sketch_update_start = std::chrono::steady_clock::now();
  gpuSketchTest_kernel<<<num_device_blocks, num_device_threads, maxBytes>>>(num_device_blocks, num_nodes, num_updates_per_blocks, sketchParams.num_buckets, d_buckets, sketchParams.num_columns, sketchParams.bkt_per_col, sketchSeed);
  hipDeviceSynchronize();
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) 
      printf("Error: %s\n", hipGetErrorString(err));
  std::chrono::duration<double> sketch_update_duration = std::chrono::steady_clock::now() - sketch_update_start;

  std::cout << "Total insertion time(sec):    " << sketch_update_duration.count() << std::endl;
  std::cout << "Updates per second:           " << num_updates / sketch_update_duration.count() << std::endl;

  hipFree(d_buckets);
}
