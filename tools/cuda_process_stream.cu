#include "hip/hip_runtime.h"
#include <vector>
#include <graph.h>
#include <graph_worker.h>
#include <map>
#include <binary_graph_stream.h>
#include <cuda_graph.cuh>

int main(int argc, char **argv) {
  if (argc != 4) {
    std::cout << "ERROR: Incorrect number of arguments!" << std::endl;
    std::cout << "Arguments: stream_file, graph_workers, reader_threads" << std::endl;
    exit(EXIT_FAILURE);
  }

  std::string stream_file = argv[1];
  int num_threads = std::atoi(argv[2]);
  if (num_threads < 1) {
    std::cout << "ERROR: Invalid number of graph workers! Must be > 0." << std::endl;
    exit(EXIT_FAILURE);
  }
  int reader_threads = std::atoi(argv[3]);

  BinaryGraphStream_MT stream(stream_file, 1024*32);
  node_id_t num_nodes = stream.nodes();
  size_t num_updates  = stream.edges();
  std::cout << "Running process_stream with CUDA: " << std::endl;
  std::cout << "Processing stream: " << stream_file << std::endl;
  std::cout << "nodes       = " << num_nodes << std::endl;
  std::cout << "num_updates = " << num_updates << std::endl;
  std::cout << std::endl;

  CudaGraph cudaGraph;

  auto config = GraphConfiguration().gutter_sys(CACHETREE).num_groups(num_threads);
  config.gutter_conf().gutter_factor(-4);
  Graph g{num_nodes, config, &cudaGraph, reader_threads};

  Supernode** supernodes;
  supernodes = g.getSupernodes();

  // Get variable from sample supernode
  int num_sketches = supernodes[0]->get_num_sktch();
  
  // Get variables from sample sketch
  size_t num_elems = supernodes[0]->get_sketch(0)->get_num_elems();
  size_t num_columns = supernodes[0]->get_sketch(0)->get_columns();
  size_t num_guesses = supernodes[0]->get_sketch(0)->get_num_guesses();

  std::cout << "num_sketches: " << num_sketches << "\n";
  std::cout << "num_elems: " << num_elems << "\n";
  std::cout << "num_columns: " << num_columns << "\n";
  std::cout << "num_guesses: " << num_guesses << "\n";

  // Start timer for initializing
  auto init_start = std::chrono::steady_clock::now();
  
  CudaUpdateParams* cudaUpdateParams;
  gpuErrchk(hipMallocManaged(&cudaUpdateParams, sizeof(CudaUpdateParams)));
  cudaUpdateParams[0] = CudaUpdateParams(num_nodes, num_updates, num_sketches, num_elems, num_columns, num_guesses);

  for (size_t i = 0; i < num_updates * 2; i++) {
    cudaUpdateParams[0].edgeUpdates[i] = 0;
  }

  CudaSketch* cudaSketches;
  gpuErrchk(hipMallocManaged(&cudaSketches, num_nodes * num_sketches * sizeof(CudaSketch)));

  long* sketchSeeds;
  gpuErrchk(hipMallocManaged(&sketchSeeds, num_nodes * num_sketches * sizeof(long)));

  // Allocate space for all buckets
  vec_t* d_bucket_a;
  vec_hash_t* d_bucket_c;
  gpuErrchk(hipMallocManaged(&d_bucket_a, (num_nodes * num_sketches * num_elems * sizeof(vec_t))));
  gpuErrchk(hipMallocManaged(&d_bucket_c, (num_nodes * num_sketches * num_elems * sizeof(vec_hash_t))));

  for (size_t i = 0; i < (num_nodes * num_sketches * num_elems); i++) {
    d_bucket_a[i] = 0;
    d_bucket_c[i] = 0;
  }

  // Create a vector of cuda supernodes and sketches
  for (int i = 0; i < num_nodes; i++) {
    for (int j = 0; j < num_sketches; j++) {
      Sketch* sketch = supernodes[i]->get_sketch(j);

      int bucket_id = (i * num_sketches * num_elems) + (j * num_elems);
      vec_t* bucket_a = &d_bucket_a[bucket_id];
      vec_hash_t* bucket_c = &d_bucket_c[bucket_id];

      // Rewrite sketch's bucket_a and bucket_c memory location
      sketch->set_bucket_a(bucket_a);
      sketch->set_bucket_c(bucket_c);

      CudaSketch cudaSketch(bucket_a, bucket_c, sketch->get_seed());
      cudaSketches[(i * num_sketches) + j] = cudaSketch;
      sketchSeeds[(i * num_sketches) + j] = sketch->get_seed();
    }
  }

  int device_id = hipGetDevice(&device_id);
  int device_count = 0;
  hipGetDeviceCount(&device_count);
  std::cout << "CUDA Device Count: " << device_count << "\n";
  std::cout << "CUDA Device ID: " << device_id << "\n";

  std::cout << "Allocated Shared Memory of: " << (num_elems * num_sketches * sizeof(vec_t_cu)) + (num_elems * num_sketches * sizeof(vec_hash_t)) << "\n";

  // Prefetch memory to device 
  gpuErrchk(hipMemPrefetchAsync(cudaSketches, num_nodes * num_sketches * sizeof(CudaSketch), device_id));
  gpuErrchk(hipMemPrefetchAsync(sketchSeeds, num_nodes * num_sketches * sizeof(long), device_id));
  gpuErrchk(hipMemPrefetchAsync(d_bucket_a, num_nodes * num_sketches * num_elems * sizeof(vec_t), device_id));
  gpuErrchk(hipMemPrefetchAsync(d_bucket_c, num_nodes * num_sketches * num_elems * sizeof(vec_hash_t), device_id));

  cudaGraph.configure(cudaUpdateParams, cudaSketches, sketchSeeds, num_threads);
  
  GutteringSystem *gts = g.getGTS();

  std::cout << "Finished initializing CUDA parameters\n";
  std::chrono::duration<double> init_time = std::chrono::steady_clock::now() - init_start;
  std::cout << "CUDA parameters init duration: " << init_time.count() << std::endl;

  // Start timer for kernel
  auto ins_start = std::chrono::steady_clock::now();

  std::cout << "Update Kernel Starting...\n";

  int num_insertion = 10;
  std::vector<std::chrono::duration<double>> insert_durations, flush_durations;
  std::vector<std::thread> threads;
  threads.reserve(reader_threads);

  auto task = [&](const int thr_id) {
    MT_StreamReader reader(stream);
    GraphUpdate upd;
    while(true) {
      upd = reader.get_edge();
      if (upd.type == BREAKPOINT) break;
      Edge &edge = upd.edge;

      gts->insert({edge.src, edge.dst}, thr_id);
      std::swap(edge.src, edge.dst);
      gts->insert({edge.src, edge.dst}, thr_id);
    }
  };

  for (int i = 0; i < num_insertion; i++) {
    auto ins_round_start = std::chrono::steady_clock::now();
    stream.stream_reset();
    threads.clear();
    cudaGraph.reset_insertion();
    //GraphWorker::unpause_workers();
    
    /*if (i == num_insertion - 1) {
      cudaGraph.canInsert = true;
    }*/

    for (int t = 0; t < reader_threads; t++) {
      threads.emplace_back(task, t);
    } 

    for (int t = 0; t < reader_threads; t++) {
      threads[t].join();
    }  
    insert_durations.push_back(std::chrono::steady_clock::now() - ins_round_start);
    std::cout << "Insertion #" << i << " Completed\n";
    
    /*auto flush_start = std::chrono::steady_clock::now();
    gts->force_flush();
    GraphWorker::pause_workers();
    flush_durations.push_back(std::chrono::steady_clock::now() - flush_start);
    std::cout << "Force Flush #" << i << " Completed\n";*/
  }

  std::cout << "Update Kernel finished.\n";

  // End timer for kernel
  auto ins_end = std::chrono::steady_clock::now();
  
  // Update graph's num_updates value
  g.num_updates += num_updates * 2;

  // Start timer for cc
  auto cc_start = std::chrono::steady_clock::now();
  //auto CC_num = g.connected_components().size();
  auto CC_num = 0;

  std::chrono::duration<double> insert_time = ins_end - ins_start;
  std::chrono::duration<double> cc_time = std::chrono::steady_clock::now() - cc_start;
  //std::chrono::duration<double> flush_time = flush_end - flush_start;
  std::chrono::duration<double> cc_alg_time = g.cc_alg_end - g.cc_alg_start;

  double num_seconds = insert_time.count();
  std::cout << "Total insertion time(sec):    " << num_seconds << std::endl;

  for (int i = 0; i < num_insertion; i++) {
    std::cout << "  Insertion #" << i << ":                 " << insert_durations[i].count() << std::endl;
    //std::cout << "  Force Flush #" << i << ":               " << flush_durations[i].count() << std::endl;
  }

  std::cout << "Updates per second:           " << stream.edges() / num_seconds << std::endl;
  std::cout << "Total CC query latency:       " << cc_time.count() << std::endl;
  //std::cout << "  Flush Gutters(sec):           " << flush_time.count() << std::endl;
  std::cout << "  Boruvka's Algorithm(sec):     " << cc_alg_time.count() << std::endl;
  std::cout << "Connected Components:         " << CC_num << std::endl;

  /*bool first_round = true;
  int round_num = 0;

  std::vector<std::chrono::duration<double>> round_durations;
  std::vector<std::chrono::duration<double>> sample_durations;
  std::vector<std::chrono::duration<double>> to_merge_durations;
  std::vector<std::chrono::duration<double>> merge_durations;

  // Start sampling supernodes
  do {
    // Start timer for initial time for round
    auto round_start = std::chrono::steady_clock::now();
    std::cout << "Round " << round_num << "\n";

    cudaCCParams[0].modified[0] = false;

    // Number of blocks
    num_device_blocks = (cudaCCParams[0].num_nodes[0] + num_device_threads - 1) / num_device_threads;

    // Get and check sample_idx of each supernodes
    for (int i = 0; i < cudaCCParams[0].num_nodes[0]; i++) {
      int index = cudaCCParams[0].reps[i];

      if(cudaCCParams[0].sample_idxs[index] >= cudaCCParams[0].merged_sketches[index]) throw OutOfQueriesException();

      Sketch* sketch = supernodes[index]->get_sketch(cudaCCParams[0].sample_idxs[index]);

      // Check if this sketch has already been queried
      if(sketch->get_queried()) throw MultipleQueryException();
      
      sketch->set_queried(true);

      // Increment current supernode's sample idx
      cudaCCParams[0].sample_idxs[index]++;
    }

    // Start timer for sampling
    auto sample_start = std::chrono::steady_clock::now();

    // Sample each supernodes
    cuda_sample_supernodes(num_device_threads, num_device_blocks, cudaCCParams, cudaSketches);
    std::cout << "SAMPLING DONE\n";

    // End timer for sampling
    auto sample_end = std::chrono::steady_clock::now();
    sample_durations.push_back(sample_end - sample_start);

    // Start timer for to_merge
    auto to_merge_start = std::chrono::steady_clock::now();

    // Reset to_merge
    if(!first_round) {
      cudaCCParams[0].reset();
    }

    cuda_supernodes_to_merge(num_device_threads, num_device_blocks, cudaCCParams);

    std::cout << "TO_MERGE DONE\n";

    std::cout << "Reps: ";
    for (int i = 0; i < cudaCCParams[0].num_nodes[0]; i++) {
      std::cout << cudaCCParams[0].reps[i] << " ";
    }
    std::cout << "\n";

    // End timer for to_merge
    auto to_merge_end = std::chrono::steady_clock::now();
    to_merge_durations.push_back(to_merge_end - to_merge_start);

    // Start timer for merge
    auto merge_start = std::chrono::steady_clock::now();

    num_device_blocks = (cudaCCParams[0].num_nodes[0] + num_device_threads - 1) / num_device_threads;

    cuda_merge_supernodes(num_device_threads, num_device_blocks, cudaCCParams, cudaSketches);
    std::cout << "MERGE DONE\n";

    // End timer for merge
    auto merge_end = std::chrono::steady_clock::now();
    merge_durations.push_back(merge_end - merge_start);

    first_round = false;
    round_num++;

    // End timer for round
    auto round_end = std::chrono::steady_clock::now();
    round_durations.push_back(round_end - round_start);

  } while (cudaCCParams[0].modified[0]);

  for (node_id_t i = 0; i < num_nodes; ++i) {
    g.setSize(i, cudaCCParams[0].size[i]);
    g.setParent(i, cudaCCParams[0].parent[i]);
  }

  // Find connected components
  auto CC_num = g.cc_from_dsu().size();

  // End timer for cc
  auto cc_end = std::chrono::steady_clock::now();

  std::chrono::duration<double> insert_time = ins_end - ins_start;
  std::chrono::duration<double> cc_time = cc_end - cc_start;

  double num_seconds = insert_time.count();
  std::cout << "Total insertion time(sec):    " << num_seconds << std::endl;
  std::cout << "Updates per second:           " << stream.edges() / num_seconds << std::endl;
  std::cout << "Total CC query latency:       " << cc_time.count() << std::endl;

  for (int i = 0; i < sample_durations.size(); i++) {
    std::cout << "    Round " << i << ":                  " << round_durations[i].count() << std::endl;
    std::cout << "        Sampling:               " << sample_durations[i].count() << std::endl;
    std::cout << "        To_Merge:               " << to_merge_durations[i].count() << std::endl;
    std::cout << "        Merge:                  " << merge_durations[i].count() << std::endl;
  }
  std::cout << "Connected Components:         " << CC_num << std::endl;*/
}
